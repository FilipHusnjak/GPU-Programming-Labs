
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <cstdio>
#include <chrono>

#define TPB 256
#define NUM_SAMPLES 1000000
#define N 1000

#define PI 3.14159265359

__global__ void ornl(unsigned int *res, hiprandState *states) {
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= NUM_SAMPLES / N) return;

    __shared__  unsigned int count[TPB];

    hiprand_init(idx, idx, 0, &states[idx]);

    count[threadIdx.x] = 0;
    for (int i = 0; i < N; i++) {
        float x = hiprand_uniform(&states[idx]);
        float y = hiprand_uniform(&states[idx]);

        int z = (int) (x * x + y * y);
        count[threadIdx.x] += 1 - z;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        unsigned int blockSum = 0;
        for (unsigned int i = 0; i < blockDim.x; i++)
            blockSum += count[i];

        atomicAdd(res, blockSum);
    }
}

int main() {
    hiprandState *states;
    unsigned int thread_num = NUM_SAMPLES / N;
    hipMalloc(&states, thread_num * sizeof(hiprandState));

    unsigned int *d_res;
    unsigned int a = 0;
    hipMalloc(&d_res, sizeof(*d_res));
    hipMemcpy(d_res, &a, sizeof(a), hipMemcpyHostToDevice);

    auto t1 = std::chrono::system_clock::now();
    ornl<<<(thread_num + TPB - 1) / TPB, TPB>>>(d_res, states);
    hipDeviceSynchronize();
    auto t2 = std::chrono::system_clock::now();
    printf("Calculating PI on the GPU done in: %lf ms!\n",
           std::chrono::duration<double, std::chrono::milliseconds::period>(t2 - t1).count());

    unsigned int res;
    hipMemcpy(&res, d_res, sizeof(*d_res), hipMemcpyDeviceToHost);

    double pi = 4 * (double) res / (NUM_SAMPLES);
    printf("Obtained PI \t\t= %f\n", pi);
    printf("Real PI \t\t= %f\n", PI);
}