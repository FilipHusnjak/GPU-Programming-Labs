
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <cstdio>
#include <chrono>
#include <hip/hip_fp16.h>

#define TPB 256
#define NUM_SAMPLES 10000000
#define N 100000

#define PI 3.14159265359

__global__ void ornl(unsigned int *res, hiprandState *states) {
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= NUM_SAMPLES / N) return;

    __shared__  unsigned int count[TPB];

    hiprand_init(idx, idx, 0, &states[idx]);

    count[threadIdx.x] = 0;
    for (int i = 0; i < N; i++) {
        float x = hiprand_uniform(&states[idx]);
        float y = hiprand_uniform(&states[idx]);

        int z = (int) (x * x + y * y);
        count[threadIdx.x] += 1 - z;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        unsigned int blockSum = 0;
        for (unsigned int i = 0; i < blockDim.x; i++)
            blockSum += count[i];

        atomicAdd(res, blockSum);
    }
}

int main() {
    hiprandState *states;
    unsigned int thread_num = NUM_SAMPLES / N;
    hipMalloc(&states, thread_num * sizeof(hiprandState));

    unsigned int *d_res;
    unsigned int a = 0;
    hipMalloc(&d_res, sizeof(*d_res));

    auto t1 = std::chrono::system_clock::now();
    hipMemcpy(d_res, &a, sizeof(a), hipMemcpyHostToDevice);
    ornl<<<(thread_num + TPB - 1) / TPB, TPB>>>(d_res, states);
    hipDeviceSynchronize();
    unsigned int res;
    hipMemcpy(&res, d_res, sizeof(*d_res), hipMemcpyDeviceToHost);
    auto t2 = std::chrono::system_clock::now();
    printf("Calculating PI on the GPU done in: %lf ms!\n",
           std::chrono::duration<double, std::chrono::milliseconds::period>(t2 - t1).count());

    double pi = 4 * (double) res / (NUM_SAMPLES);
    printf("Obtained PI \t\t= %f\n", pi);
    printf("Real PI \t\t= %f\n", PI);
}
